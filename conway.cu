#include <stdio.h>
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
// #define PRINT // use with ROW_ELEMENTS < 50

// Cells definitions
#define ALIVE 35 // #
#define DEAD 32 // (space)
#define BORDER 66 // B

// Kernel definition
#define THREADS_PER_BLOCK 256
#define BLOCKS_PER_GRID 1

// Universe definition
#define STEPS 10
#define ROW_ELEMENTS 1000 // 3000 1x1 = 42s
#define ROW_WITH_BORDER_ELEMENTS (ROW_ELEMENTS + 2)

__device__ int getNeighboursCount(const char *universe, int i) {
    int count = 0;
    int startIndex = i - ROW_WITH_BORDER_ELEMENTS - 1;
    for (int j = 0; j < 3; j++) {
        for (int k = 0; k < 3; k++) {
            int currentIndex = startIndex + k;
            if (currentIndex != i && // not itself
                universe[currentIndex] == ALIVE) {
                count++;
            }
        }
        startIndex += ROW_WITH_BORDER_ELEMENTS;
    }
    return count;
}

__global__ void computeConwayUniverse(const char *in_universe, char *out_universe, long long int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    for (; i < numElements; i += blockDim.x * gridDim.x) {
        if (in_universe[i] == BORDER) {
            out_universe[i] = BORDER;
        } else {
            int neighboursCount = getNeighboursCount(in_universe, i);
            out_universe[i] =
                    neighboursCount == 3 ? ALIVE : neighboursCount == 2 && in_universe[i] == ALIVE ? ALIVE : DEAD;
        }
    }
}

void checkError(hipError_t err, const char *format) {
    if (err != hipSuccess) {
        fprintf(stderr, format, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main(void) {
    srand(time(NULL));
    clock_t start, end;

    hipError_t err = hipSuccess;

    long long int numElements = (long long int) ROW_WITH_BORDER_ELEMENTS * (long long int) ROW_WITH_BORDER_ELEMENTS;
    size_t universe_size = numElements * sizeof(char);

    printf("[Universe of size %d x %d (%d blocks with %d threads, %d steps)]\n", ROW_ELEMENTS, ROW_ELEMENTS, BLOCKS_PER_GRID, THREADS_PER_BLOCK, STEPS);

    char *h_universe = (char *) malloc(universe_size);
    if (h_universe == NULL) {
        fprintf(stderr, "Failed to allocate host h_universe!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize universe
    for (int i = 0; i < numElements; ++i) {
        if (i < ROW_WITH_BORDER_ELEMENTS ||  // first row
            i >= numElements - ROW_WITH_BORDER_ELEMENTS || // last row 
            i % ROW_WITH_BORDER_ELEMENTS == 0 || // first column
            i != 0 && i % ROW_WITH_BORDER_ELEMENTS == ROW_WITH_BORDER_ELEMENTS - 1) { // last column
            h_universe[i] = BORDER;
        } else {
            h_universe[i] = rand() % 2 == 0 ? DEAD : ALIVE;
        }
    }

    char *d_in_universe = NULL;
    err = hipMalloc((void **) &d_in_universe, universe_size);
    checkError(err, "Failed to allocate device universe (error code %s)!\n");

    char *d_out_universe = NULL;
    err = hipMalloc((void **) &d_out_universe, universe_size);
    checkError(err, "Failed to allocate device universe (error code %s)!\n");

    start = clock();
    for (int i = 0; i < STEPS; i++) {
        err = hipMemcpy(d_in_universe, h_universe, universe_size, hipMemcpyHostToDevice);
        checkError(err, "Failed to copy universe from host to device (error code %s)!\n");

        computeConwayUniverse<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(d_in_universe, d_out_universe, numElements);

        err = hipGetLastError();
        checkError(err, "Failed to launch Conway kernel (error code %s)!\n");

        err = hipMemcpy(h_universe, d_out_universe, universe_size, hipMemcpyDeviceToHost);
        checkError(err, "Failed to copy universe from device to host (error code %s)!\n");

#ifdef PRINT
        printf("\e[0;1H\e[2J");
        printf("Step (%d) CUDA Conway kernel launch with %d blocks of %d threads\n", i, BLOCKS_PER_GRID, THREADS_PER_BLOCK);
        for (int i=0; i < numElements; ++i) {
            if (i % ROW_WITH_BORDER_ELEMENTS == 0){
                printf("\n");
            }
                printf("%c ", h_universe[i]);
        }
        printf("\n");
        sleep(1);
#endif
    }
    end = clock();
    printf("Computations took %f s\n", ((double) (end - start) / CLOCKS_PER_SEC));

    err = hipFree(d_in_universe);
    checkError(err, "Failed to free device in universe (error code %s)!\n");

    err = hipFree(d_out_universe);
    checkError(err, "Failed to free device out universe (error code %s)!\n");

    free(h_universe);

    err = hipDeviceReset();
    checkError(err, "Failed to deinitialize the device! error=%s\n");

    return 0;
}
